#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <fstream>
#include <string>
#include <sstream>
#include <chrono>
#include <omp.h>
#include <algorithm>
#include <cassert>

using namespace std;
using us = std::chrono::microseconds;

auto constexpr BLOCK_SIZE = 256u;

bool check_empty(unsigned short* re_color, int vert_num){
    auto count = 0;
    for (int i=0; i<vert_num; i++){
         count+=re_color[i];
    }

    return count == 0;
}

void print_coloring(unsigned short* colors, int vert_num) {
    for (int j=0;j<vert_num;j++){
        cout<<"Vertex"<<j<<" ---> Color: "<<colors[j]<<endl;
    }
}

void print_adj_list(vector<int> vertices[], int vert_num){
    for(int i=0; i<vert_num;i++){
        vector<int> curr = vertices[i];
        cout<<"Vertex:"<<i<<endl;
        for (auto j = 0lu; j < curr.size(); j++)
        {
            cout<<curr[j]<<" ";
        }
        cout<<endl;
    }
}

void better_coloring (int vert_num, vector<unsigned short> vertices []) {
    unsigned short* colors = new unsigned short[vert_num]();
    unsigned short* re_color = new unsigned short[vert_num]();
    for (int i=0; i<vert_num; i++){
        re_color[i] = 1;
    }
    while(!check_empty(re_color, vert_num)){
        #pragma omp parallel for
        for (int i=0; i<vert_num;i++){
            if (re_color[i] == 1){
                auto const ad_len = vertices[i].size();
                short* f_col = new short[vert_num];
                for(int j=0;j<vert_num;j++){
                    f_col[j] = -1;
                }
                
                for (auto j=0lu; j<ad_len; j++){
                    auto& adj_index = vertices[i][j];
                    f_col[colors[adj_index]]=i;
                }
                unsigned short j=0;
                bool stop = false;
                //cout<<i<<endl;
                while(!stop){
                    stop = (f_col[j] != i);
                    j++;
                }
                colors[i] = j-1;
                re_color[i] = 0;
                delete [] f_col;
            }  
        }
        /*
        for (int i =0; i<vert_num;i++){
            cout<<re_color[i]<<" ";
        }
        */
        //cout<<endl;

        #pragma omp parallel for
        for (int i=0; i<vert_num;i++){   
            auto const ad_len = vertices[i].size();
            for (auto j=0lu; j<ad_len; j++){
                auto& adj_index = vertices[i][j];
                if ((colors[i] == colors[adj_index]) && (i > adj_index)){
                    re_color[i] = 1;
                    //cout<<"entra"<<endl;
                }
            }
        }

    }
    delete [] re_color;
    print_coloring(colors, vert_num);
    cout<<"-------------"<<endl;
    delete [] colors;
}

__global__
void parallel_coloring(unsigned short * out, vector<unsigned short> *dev_data, size_t vert) {
    auto const th_id =  blockIdx.x * blockDim.x + threadIdx.x;
    if (th_id == 0) {
        for (int i=0; i<vert;i++){
            cout<<dev_data[i][0]<<endl;
        }
    }
}

int main( int argc, char ** argv) {

    string fname = "test_graph_2.csv";
    unsigned short const vert_num = 20;
    vector<unsigned short> vertices [vert_num];
    for (int i=0; i<vert_num;i++){
        vector<unsigned short> temp;
        vertices[i] = temp;
    }
    string line, word;
    vector<string> row;
    fstream file (fname, ios::in);
    if(file.is_open()) {
        while(getline(file, line)) {
            row.clear();
            stringstream str(line);
            while(getline(str, word, ',')){
                row.push_back(word);
            }
            if (isdigit(row[0][0])){
                unsigned short v1 = stoi(row[0]);
                unsigned short v2 = stoi(row[1]);
                vertices[v1].push_back(v2);
                vertices[v2].push_back(v1);
            }
        }
    }
    else {
        cout<<"Could not open the file\n";
    }

    unsigned short colors[vert_num]; 

    vector<unsigned short> *dev_data;
    unsigned short *dev_output;
    hipMalloc((void**) &dev_data, sizeof(void *) * vert_num);
    hipMalloc((void**) &dev_output, sizeof(unsigned short)*vert_num);

    hipMemcpy( dev_data, vertices, sizeof(void *)*vert_num, hipMemcpyHostToDevice);

    parallel_coloring<<< 1, BLOCK_SIZE >>>( dev_output, dev_data, vert_num);

    hipMemcpy( colors, dev_output, sizeof(unsigned short)*vert_num, hipMemcpyDeviceToHost);
    
    /*
    auto const benchmark_trials = 10000;
    auto const start_time = std::chrono::system_clock::now();
    for( int i = 0; i < benchmark_trials; i++ )
        better_coloring(vert_num, vertices);
    auto const end_time = std::chrono::system_clock::now();
    auto elapsed_time = std::chrono::duration_cast< us >( end_time - start_time );
    std::cout << "average time per run: "
            << elapsed_time.count() / static_cast< float >( benchmark_trials )
            << " us" << std::endl;
    */
    
    return 0;
}